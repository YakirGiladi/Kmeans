#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "main_functions.h"
#include <hip/device_functions.h>
#include <thrust\device_vector.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

__global__ void calc2points(float* point_coordinate_1, float* point_coordinate_2 , float* coordinates_arr);
hipError_t calcDistanceCoordiantesWithCuda(float* coordinates_1, float* coordinates_2, float* coordinates_arr, int num_coordinates);

void print_vars(int N, int n,int MAX,int LIMIT, float QM);
void error(void* void_1 , void* void_2, void* void_3);

int main()
{
	clock_t begin = clock();

	int k = 2; // init number of cluster
	int N,n,MAX,LIMIT; // 250, 52, 30, 200
	float QM; // 7.3
	float* arr_var = (float*)malloc(sizeof(float)*5);

	arr_var = getVarsFromTXT();
	N = (int)arr_var[0]; // Number of point
	n = (int)arr_var[1]; // Number of coordinates
	MAX = (int)arr_var[2]; 
	LIMIT = (int)arr_var[3]; // LIMIT iterations
	QM = arr_var[4]; 
	print_vars(N,n,MAX,LIMIT,QM);
	Point* all_points = (Point*)malloc(sizeof(Point)*N);
	all_points = getPointsFromTXT(all_points,N,n); // get Points
	Cluster* all_clusters = (Cluster*)calloc(k, sizeof(Cluster)); // Create Clusters

	//print_number_of_points_cluster(all_clusters, k);
	//print_diamters(all_clusters,k);

	all_clusters = init_clusters(all_clusters,all_points, k, N); // initiate clusters

	printf("Iterations Starts:\n----------------------\n");

	while(k < MAX) {

		printf("Number of Clusters = %d\n",k);
		for (int i = 0 ; i < LIMIT ; i++)
		{
			printf("\nIteration %d:\n------------------\n",i);

			associate_points_to_clusters(all_points, all_clusters, N, k, n); // Associate points to clusters
			print_cluster_ditails(all_clusters, k);
			printf("Recenter clusters ..\n");
			recenter_all_clusters(all_clusters, k , n); // Recenter Clusters
			
			//print_clusters_centers(all_clusters, k , n);
			//system("pause");
			if(check_termination_condition(all_clusters, k, n))
			{
				printf("Termination condition Happened\n\n");
				i = LIMIT;
			}
			else
			{
				reset_points_and_diameter_in_clusters(all_clusters, k);
				//associate_points_to_clusters(all_points, all_clusters, N, k, n); // Associate points to clusters checking
			}
		}

		double new_quality = calc_qm(all_clusters,k, n);
		//double new_quality = calculateQM(all_clusters,k,n);

		printf("new_quality:%f\n",new_quality);

		if(new_quality <= QM)
		{	
			clock_t end = clock();
			double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

			printf("\nFinished with good quality = %f\n", new_quality);
			printf("Number of clusters: %d\n",k);

			print_cluster_ditails(all_clusters, k);

			if (parallel)
				printf("In Parallel:\n");
			else
				printf("In Sequence:\n");
			printf("Time Spent: %f Seconds\n",time_spent);

			write_results_to_file(all_clusters, k, n, new_quality);

			free(all_points);
			free(all_clusters);
			break;
		}
		else 
		{
			printf("The new QM is bigger than %.2f\n",QM);
			printf("Increase the number of clusters\n\n");
			k++;
			free(all_clusters);
			all_clusters = (Cluster*)calloc(k, sizeof(Cluster));
			all_clusters = init_clusters(all_clusters,all_points, k, N); // initiate clusters
		}
	}
}

void print_vars(int N, int n,int MAX,int LIMIT, float QM)
{
	printf("Variables:\n------------\n");
	printf("N = %d Products\n",N);
	printf("n = %d Coordinates\n",n);
	printf("MAX = %d Maximum Clusters\n",MAX);
	printf("LIMIT = %d Limit Iterations\n",LIMIT);
	printf("QM = %.2f Quality Clusters\n\n",QM);
}

void error(void* void_1 , void* void_2, void* void_3)
{
	hipFree(void_1);
	hipFree(void_2);
	hipFree(void_3);
}


__global__ void calc2points(float* point_coordinate_1, float* point_coordinate_2 , float* coordinates_arr)
{
    int tid = threadIdx.x; // 52

	coordinates_arr[tid] = pow(point_coordinate_1[tid] - point_coordinate_2[tid],2);
}


 //Helper function for using CUDA to add vectors in parallel.
hipError_t calcDistanceCoordiantesWithCuda(float* coordinates_1, float* coordinates_2, float* coordinates_arr, int num_coordinates)
{
    float* dev_coordinates_1;
    float* dev_coordinates_2;
	float* dev_coordinates_arr;

	dev_coordinates_1 = (float*)malloc(sizeof(float)*num_coordinates);
	dev_coordinates_2 = (float*)malloc(sizeof(float)*num_coordinates);
	dev_coordinates_arr = (float*)malloc(sizeof(float)*num_coordinates);
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        //goto Error;
		error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
    }

    // Allocate GPU buffers for three vectors (two input, one output)
	// dev_coordinates_1
	cudaStatus = hipMalloc((void**)&dev_coordinates_1, sizeof(float)*num_coordinates);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc of dev_coordinates_1 failed!\n");
        //goto Error;
		error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
    }

	// dev_coordinates_2
    cudaStatus = hipMalloc((void**)&dev_coordinates_2, sizeof(float)*num_coordinates);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc of dev_coordinates_2 failed!\n");
        //goto Error;
		error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
    }

	// coordinates_arr
	cudaStatus = hipMalloc((void**)&dev_coordinates_arr, sizeof(float)*num_coordinates);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc of dev_coordinates_arr failed!\n");
        //goto Error;
		error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
    }

	
    // Copy from host memory to GPU buffers.
	cudaStatus = hipMemcpyAsync(dev_coordinates_1, coordinates_1, sizeof(float)*num_coordinates, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyAsync dev_coordinates_1 failed!");
        //goto Error;
		error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
    }

	cudaStatus = hipMemcpyAsync(dev_coordinates_2, coordinates_2, sizeof(float)*num_coordinates, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyAsync dev_coordinates_2 failed!");
        //goto Error;
		error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
    }

	cudaStatus = hipMemcpyAsync(dev_coordinates_arr, coordinates_arr, sizeof(float)*num_coordinates, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyAsync dev_coordinates_arr failed!");
		printf("stderr:%s\n",stderr);
        //goto Error;
		error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
    }

    // Launch a kernel on the GPU with one thread for each element.
	calc2points<<<1, 52>>>(dev_coordinates_1, dev_coordinates_2 ,dev_coordinates_arr);
	//calc2pointsWith4Blocks<<<4, 13>>>(dev_coordinates_1, dev_coordinates_2 ,dev_coordinates_arr);
    // Check for any errors launching the kernel
    
	cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        //goto Error;
		error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        //goto Error;
		error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpyAsync(coordinates_arr, dev_coordinates_arr, sizeof(float)*num_coordinates, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        //goto Error;
		error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
    }
	error(dev_coordinates_1, dev_coordinates_2, dev_coordinates_arr);
	return cudaStatus;
}